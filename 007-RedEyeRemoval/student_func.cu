//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

#define USE_THRUST 0


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //PUT YOUR SORT HERE

	/* Explore the data
	std::cout << "numElems " << numElems << std::endl; //220480

	unsigned int* h_inputVals = new unsigned int[numElems];

	checkCudaErrors(hipMemcpy(h_inputVals, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));

	for (int cnt = 0; cnt < 100; ++cnt)
	{
		std::cout << h_inputVals[cnt] << " " ;
		if ((cnt + 1)%10 == 0)
		{
			std::cout << std::endl;
		}
	}

	delete[] h_inputVals;
	*/

#if USE_THRUST
	thrust::device_vector<unsigned int> d_keys(d_inputVals, d_inputVals + numElems);
	thrust::device_vector<unsigned int> d_values(d_inputPos, d_inputPos + numElems);

	thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_values.begin());

	thrust::copy(d_keys.begin(), d_keys.end(), thrust::device_pointer_cast(d_outputVals));
	thrust::copy(d_values.begin(), d_values.end(), thrust::device_pointer_cast(d_outputPos));


#else
	int num_bits = sizeof(unsigned int) * 8;

#endif
}
